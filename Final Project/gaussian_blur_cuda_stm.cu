#include "hip/hip_runtime.h"
#include "bmpReader.h"
#include "bmpReader.cpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <bits/stdc++.h>
#include <sys/time.h>

using namespace std;

// CUDA Stream
#define N_STREAMS 8

// Gaussian filter
int filter_size;
unsigned int filter_scale, filter_row;
unsigned int *filter;

// Image IO
int img_col, img_row;
unsigned char *img_input, *img_output;

// CUDA error checker
void cuda_err_chk(const hipError_t& e, const int& cudaError_cnt){
    if(e != hipSuccess){
        fprintf(stderr, "hipError_t in no. %d: %s\n", cudaError_cnt, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
}


// Kernel
__global__ void cuda_gaussian_filter(unsigned char* img_input_cuda, unsigned char* img_output_cuda,int img_col, int img_row, int shift, unsigned int* filter_cuda, int filter_row, unsigned int filter_scale, int img_border_upper){
    int cuda_col = blockIdx.x * blockDim.x + threadIdx.x;
    int cuda_row = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int tmp = 0;
    int target = 0;
    int a, b;
    
    if (3 * (cuda_row * img_col + cuda_col) + shift >= img_border_upper){
        return;
    }
    
    for(int j = 0; j < filter_row; j++){
        for(int i = 0; i < filter_row; i++){
            a = cuda_col + i - (filter_row / 2);
            b = cuda_row + j - (filter_row / 2);

            target = 3 * (b * img_col + a) + shift;
            if (target >= img_border_upper || target < 0){
                continue;
            }

			tmp += filter_cuda[j * filter_row + i] * img_input_cuda[target];  
        }
    }
    tmp /= filter_scale;

    if(tmp > 255){
        tmp = 255;
    }
    
    img_output_cuda[3 * (cuda_row * img_col + cuda_col) + shift] = tmp;
}

int main(int argc, char* argv[])
{
    /*--------------- Init -------------------*/
    int thread_cnt, block_row;
    int cudaError_cnt = 0;
    string fname_input;
    string fname_output;

    if(argc < 2){
        fprintf(stderr, "%s", "Please provide filename for Gaussian Blur. usage ./gb_std.o <BMP image file> \n");
        return -1;
    }
    else if(argc == 3){
        sscanf(argv[2], "%d", &thread_cnt);
        printf("Testing with %d threads in each CUDA block\n", thread_cnt);
    }
    else{
        // Set default thread count to 1024
        thread_cnt = 1024;
    }
    block_row = (int)sqrt(thread_cnt);

    /*---------------- Image and mask IO ----*/
    FILE* mask;
    mask = fopen("mask_Gaussian.txt", "r");
    fscanf(mask, "%d", &filter_size);
    filter_row = (int)sqrt(filter_size);
    filter = new unsigned int [filter_size];

    for(int i = 0; i < filter_size; i++){
        fscanf(mask, "%u", &filter[i]);
    }

    filter_scale = 0;
    for(int i = 0; i < filter_size; i++){
        filter_scale += filter[i];	
    }
    fclose(mask);

    
    /*-------------- CUDA init ------------*/
    int num = 0;
    hipGetDeviceCount(&num);
    hipDeviceProp_t prop;
    if(num > 0){
        hipGetDeviceProperties(&prop, 0);
        cout << "Device: " << prop.name << '\n';
    }
    else{
        fprintf(stderr, "%s", "No NVIDIA GPU detected!\n");
        return 1;
    }
    
    fname_input = argv[1];
    BmpReader* bmp_io = new BmpReader();
    img_input = bmp_io->ReadBMP(fname_input.c_str(), &img_col, &img_row);
    printf("Filter scale = %u, filter size %d x %d and image size W = %d, H = %d\n", filter_scale, filter_row, filter_row, img_col, img_row);

    int resolution = 3 * (img_col * img_row);
    img_output = (unsigned char*)malloc(resolution * sizeof(unsigned char));
    memset(img_output, 0, sizeof(img_output));
    // Apply the Gaussian filter to the image, RGB respectively
    string tmp(fname_input);

    // Init CUDA streams
    hipStream_t streams[N_STREAMS];
    for (int i = 0; i < N_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }
    int offset = 0;
    int chunk_size = resolution / N_STREAMS;

    unsigned char* img_input_cuda;
    unsigned char* img_output_cuda;
    unsigned int* filter_cuda;
    cuda_err_chk(hipMalloc((void**) &img_input_cuda, resolution * sizeof(unsigned char)), cudaError_cnt++);
    cuda_err_chk(hipMalloc((void**) &img_output_cuda, resolution * sizeof(unsigned char)), cudaError_cnt++);
    cuda_err_chk(hipMalloc((void**) &filter_cuda, filter_size * sizeof(unsigned int)), cudaError_cnt++);
    
    // Copy memory from host to GPU
    cuda_err_chk(hipMemcpy(filter_cuda, filter, filter_size * sizeof(unsigned int), hipMemcpyHostToDevice), cudaError_cnt++); // The filter matrix with constant memory
    
    // Grid and block, divide the image into 1024 per block
    /* 
    Split into 8 parts
    [
        _
        _
        _
        _
        _
        _
        _
        _
    ]   
    */
    const dim3 block_size(block_row, block_row);
    // const dim3 grid_size((img_col + block_row - 1) / block_row, (img_row + block_row - 1) / block_row / N_STREAMS); // BUG BAD VERSION, CAUSE ZEBRA !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    const dim3 grid_size((img_col + block_row - 1) / block_row, (img_row / N_STREAMS + block_row - 1) / (block_row));
    
    /*-------------- CUDA run ------------*/
    // R G B channel respectively
    struct timeval start, end; 
    gettimeofday(&start, 0);
    for(int j = 0; j < N_STREAMS; j++){
        offset = chunk_size * j;
        
        cuda_err_chk(hipMemcpyAsync(img_input_cuda + offset, img_input + offset, chunk_size * sizeof(unsigned char), hipMemcpyHostToDevice, streams[j]), cudaError_cnt++);
        cuda_err_chk(hipMemcpyAsync(img_output_cuda + offset, img_output + offset, chunk_size * sizeof(unsigned char), hipMemcpyHostToDevice, streams[j]), cudaError_cnt++);

        for(int i = 0; i < 3; i++) {
            cuda_gaussian_filter<<<grid_size, block_size, 0, streams[j]>>>(img_input_cuda + offset, img_output_cuda + offset, img_col, img_row, i, filter_cuda, filter_row, filter_scale, chunk_size); 
        }
        
        cuda_err_chk(hipMemcpyAsync(img_output + offset, img_output_cuda + offset, chunk_size * sizeof(unsigned char), hipMemcpyDeviceToHost, streams[j]), cudaError_cnt++);
    }
    cuda_err_chk(hipDeviceSynchronize(), cudaError_cnt++);
    gettimeofday(&end, 0);
    int sec = end.tv_sec - start.tv_sec;
    int usec = end.tv_usec - start.tv_usec;
    int t_gpu = sec * 1000 + (usec / 1000);
    printf("GPU time (ms): %d\n", t_gpu);
    
    /*-------------- Finalize ------------*/
    // Copy memory from GPU to host

    // Write output BMP file
    fname_output = fname_input.substr(0, fname_input.size() - 4)+ "_blur_cuda_stm.bmp";
    bmp_io->WriteBMP(fname_output.c_str(), img_col, img_row, img_output);
    
    // Free memory space
    free(img_input);
    free(img_output);
    free(filter);
    cuda_err_chk(hipFree(img_input_cuda), cudaError_cnt++);
    cuda_err_chk(hipFree(img_output_cuda), cudaError_cnt++);
    cuda_err_chk(hipFree(filter_cuda), cudaError_cnt++);

    printf("Finished \n");
    return 0;
}