#include "hip/hip_runtime.h"
#include "bmpReader.h"
#include "bmpReader.cpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <pthread.h>
#include <string>

// openCV libraries for showing the images dont change
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

#define MYRED	2
#define MYGREEN 1
#define MYBLUE	0
#define RATE 1000000
int img_width, img_height;

int FILTER_SIZE;
float FILTER_SCALE;
float *filter_G;

unsigned char *input_image, *pic_blur, *output_image;

// variables for cuda parallel processing
int TILE_WIDTH;

__global__ void cuda_gaussian_filter(unsigned char* input_image, unsigned char* output_image,int img_width, int img_height, int shift, float* filter_G, int ws, float FILTER_SCALE)
{
    // for CUDA parallelization
    int cuda_width = blockIdx.x * blockDim.x + threadIdx.x;
    int cuda_height = blockIdx.y * blockDim.y + threadIdx.y;
    if (cuda_width >= img_width || cuda_height >= img_height)
    {
        return;
    }
    // printf("cuda_width = %d, cuda_height = %d ", cuda_width, cuda_height);
    int tmp = 0;
    int a, b;
    for (int j = 0; j < ws; j++)
    {
        for (int i = 0; i < ws; i++)
        {
            a = cuda_width + i - (ws / 2);
            b = cuda_height + j - (ws / 2);
            // detect for borders of the image
            if (a < 0 || b < 0 || a >= img_width || b >= img_height)
            {
                continue;
            }
            //printf("Location = %d \n", 3 * (b * img_width + a) + shift);
            printf(" j * ws + i = %d j = %d i = %d\n", j * ws + i, j, i);
            printf("process w = %d h = %d  ws = %d\n", a, b, ws);
            tmp += filter_G[j * ws + i] * input_image[3 * (b * img_width + a) + shift];
            // printf(" , Value = %d ", input_image[3 * (b * img_width + a) + shift]);
            // printf("\n");
            // printf(" i = %d j = %d \n", i, j);
        }
    }

    tmp /= FILTER_SCALE;

    if (tmp < 0)
    {
        tmp = 0;
    } 
    if (tmp > 255)
    {
        tmp = 255;
    }

    output_image[3 * (cuda_height * img_width + cuda_width) + shift] = tmp;
}
// show the progress of gaussian segment by segment
const float segment[] = { 0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f, 0.9f, 1.0f };
void write_and_show(BmpReader* bmpReader, string outputblur_name, int k)
{

    // write output BMP file
    outputblur_name = "input" + to_string(k) + "_blur.bmp";
    bmpReader->WriteBMP(outputblur_name.c_str(), img_width, img_height, output_image);

    // show the output file
    Mat img = imread(outputblur_name);
    imshow("Current progress", img);
    waitKey(20);
}

int main(int argc, char* argv[])
{
    TILE_WIDTH = 1024;

    // read input filename
    string inputfile_name;
    string outputblur_name;

    if (argc < 2)
    {
        printf("Please provide filename for Gaussian Blur. usage ./gb_std.o <BMP image file>");
        return 1;
    }

    // read Gaussian mask file from system
    FILE* mask;
    mask = fopen("mask_Gaussian.txt", "r");
    fscanf(mask, "%d", &FILTER_SIZE);
    filter_G = new float[FILTER_SIZE];

    for (int i = 0; i < FILTER_SIZE; i++)
    {
        fscanf(mask, "%f", &filter_G[i]);
    }

    FILTER_SCALE = 0.0f; //recalculate
    for (int i = 0; i < FILTER_SIZE; i++)
    {
        filter_G[i] *= RATE;
        FILTER_SCALE += filter_G[i];	
    }
    fclose(mask);

    // main part of Gaussian blur
    BmpReader* bmpReader = new BmpReader();

    for (int k = 1; k < argc; k++)
    {

        // read input BMP file
        inputfile_name = argv[k];
        input_image = bmpReader -> ReadBMP(inputfile_name.c_str(), &img_width, &img_height);
        printf("Filter scale = %f and image size W = %d, H = %d\n", FILTER_SCALE, img_width, img_height);

        // allocate space for output image
        int resolution = 3 * (img_width * img_height); //padding
        output_image = (unsigned char*)malloc(resolution * sizeof(unsigned char));
        memset(output_image, 0, sizeof(output_image));
        // apply the Gaussian filter to the image, RGB respectively
        string tmp(inputfile_name);

        //---------------------CUDA main part-------------------------//
        // allocate space
        hipError_t cuda_err;

        unsigned char* cuda_input_image;
        unsigned char* cuda_output_image;

        cuda_err = hipMalloc((void**) &cuda_input_image, resolution * sizeof(unsigned char));
        cuda_err = hipMalloc((void**) &cuda_output_image, resolution * sizeof(unsigned char));
        if(cuda_err != hipSuccess)
        {
            printf("Failed with error %s \n", hipGetErrorString(cuda_err));
        }

        // copy memory from host to GPU
        hipMemcpy(cuda_input_image, input_image, resolution * sizeof(unsigned char), hipMemcpyHostToDevice);
        hipMemcpy(cuda_output_image, output_image, resolution * sizeof(unsigned char), hipMemcpyHostToDevice);
        if(cuda_err != hipSuccess)
        {
            printf("Failed with error %s \n", hipGetErrorString(cuda_err));
        }


        for (int i = 2; i >= 0; i++) //R G B channel respectively
        {
            cuda_gaussian_filter<<<(resolution + TILE_WIDTH) / TILE_WIDTH, TILE_WIDTH>>>(cuda_input_image, cuda_output_image, img_width, img_height, i, filter_G, (int)sqrt((int)FILTER_SIZE), FILTER_SCALE);
            hipError_t cuda_err = hipDeviceSynchronize();

            if(cuda_err != hipSuccess)
            {
                printf("Failed with error %s \n", hipGetErrorString(cuda_err));
                return -1;
            }
        }

        // copy memory from GPU to host
        hipMemcpy(output_image, cuda_output_image, resolution * sizeof(unsigned char), hipMemcpyDeviceToHost);
        //---------------------CUDA main part-------------------------//

        // write output BMP file
        outputblur_name = inputfile_name.substr(0, inputfile_name.size() - 4)+ "_blur.bmp";
        bmpReader->WriteBMP(outputblur_name.c_str(), img_width, img_height, output_image);
        /*Mat img = imread(outputblur_name);
          while(1)
          {
          imshow("Current progress", img);
          if(waitKey(10) == 27)
          {
          break;
          }
          }*/

        // free memory space
        free(input_image);
        free(output_image);
        hipFree(cuda_input_image);
        hipFree(cuda_output_image);
    }

    return 0;
}
