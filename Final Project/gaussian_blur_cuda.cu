#include "hip/hip_runtime.h"
#include "bmpReader.h"
#include "bmpReader.cpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <pthread.h>
#include <string>

// openCV libraries for showing the images dont change
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

#define ull unsigned long long int

#define MYRED	2
#define MYGREEN 1
#define MYBLUE	0
int img_width, img_height;

int FILTER_SIZE;
ull FILTER_SCALE;
ull *filter_G;

unsigned char *input_image, *pic_blur, *output_image;

// variables for cuda parallel processing
int TILE_WIDTH;

__global__ void cuda_gaussian_filter(unsigned char* cuda_input_image, unsigned char* cuda_output_image,int img_width, int img_height, int shift, unsigned long long int* cuda_filter_G, int ws, unsigned long long int FILTER_SCALE, int img_border)
{
    // for CUDA parallelization
    int cuda_width = blockIdx.x * blockDim.x + threadIdx.x;
    int cuda_height = blockIdx.y * blockDim.y + threadIdx.y;

    int target = 0;
    unsigned long long int tmp = 0;
    int a, b;
    
    if (3 * (cuda_height * img_width + cuda_width) + shift >= img_border)
    {
        return;
    }
    for (int j = 0; j  <  ws; j++)
	{
		for (int i = 0; i  <  ws; i++)
		{
			a = cuda_width + i - (ws / 2);
			b = cuda_height + j - (ws / 2);
            
            /* THIS CAUSE ALL PICTURE TO BE BLACK ONE
            if (a < 0 || b < 0 || a >= img_width || b >= img_height)
            {
                continue;
            }
             
             */
			// detect for borders of the image
            target = 3 * (b * img_width + a) + shift;
            if (target >= img_border || target < 0)
            {
                continue;
            }
			tmp += cuda_filter_G[j * ws + i] * cuda_input_image[3 * (b * img_width + a) + shift]; 
		}
    }
    tmp /= FILTER_SCALE;
    
    if (tmp < 0)
    {
        tmp = 0;
    } 
    if (tmp > 255)
    {
        tmp = 255;
    }
    cuda_output_image[3 * (cuda_height * img_width + cuda_width) + shift] = tmp;
    
}
// show the progress of gaussian segment by segment
const float segment[] = { 0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f, 0.9f, 1.0f };
void write_and_show(BmpReader* bmpReader, string outputblur_name, int k)
{

    // write output BMP file
    outputblur_name = "input" + to_string(k) + "_blur.bmp";
    bmpReader->WriteBMP(outputblur_name.c_str(), img_width, img_height, output_image);

    // show the output file
    Mat img = imread(outputblur_name);
    imshow("Current progress", img);
    waitKey(20);
}

int main(int argc, char* argv[])
{
    TILE_WIDTH = 1024;

    // read input filename
    string inputfile_name;
    string outputblur_name;

    if (argc < 2)
    {
        printf("Please provide filename for Gaussian Blur. usage ./gb_std.o <BMP image file>");
        return 1;
    }

    // read Gaussian mask file from system
    FILE* mask;
    mask = fopen("mask_Gaussian.txt", "r");
    fscanf(mask, "%d", &FILTER_SIZE);
	filter_G = new unsigned long long [FILTER_SIZE];

    for (int i = 0; i < FILTER_SIZE; i++)
    {
        fscanf(mask, "%llu", &filter_G[i]);
    }

    FILTER_SCALE = 0; //recalculate
    for (int i = 0; i < FILTER_SIZE; i++)
    {
        FILTER_SCALE += filter_G[i];	
    }
    fclose(mask);

    // main part of Gaussian blur
    BmpReader* bmpReader = new BmpReader();
    for (int k = 1; k < argc; k++)
    {

        // read input BMP file
        inputfile_name = argv[k];
        input_image = bmpReader -> ReadBMP(inputfile_name.c_str(), &img_width, &img_height);
        printf("Filter scale = %llu and image size W = %d, H = %d\n", FILTER_SCALE, img_width, img_height);

        // allocate space for output image
        int resolution = 3 * (img_width * img_height); //padding
        output_image = (unsigned char*)malloc(resolution * sizeof(unsigned char));
        memset(output_image, 0, sizeof(output_image));
        // apply the Gaussian filter to the image, RGB respectively
        string tmp(inputfile_name);

        //---------------------CUDA main part-------------------------//
        // allocate space
        hipError_t cuda_err, cuda_err2, cuda_err3;

        unsigned char* cuda_input_image;
        unsigned char* cuda_output_image;
        unsigned long long int* cuda_filter_G;
        cuda_err = hipMalloc((void**) &cuda_input_image, resolution * sizeof(unsigned char));
        cuda_err2 = hipMalloc((void**) &cuda_output_image, resolution * sizeof(unsigned char));
        cuda_err3 = hipMalloc((void**) &cuda_filter_G, FILTER_SIZE * sizeof(unsigned long long int)); //dont forget to allocate space for it
        if(cuda_err != hipSuccess || cuda_err2 != hipSuccess || cuda_err3 != hipSuccess)
        {
            printf("Failed with error part1 %s \n", hipGetErrorString(cuda_err));
        }

        // copy memory from host to GPU
        cuda_err = hipMemcpy(cuda_input_image, input_image, resolution * sizeof(unsigned char), hipMemcpyHostToDevice);
        cuda_err2 = hipMemcpy(cuda_output_image, output_image, resolution * sizeof(unsigned char), hipMemcpyHostToDevice);
        cuda_err3 = hipMemcpy(cuda_filter_G, filter_G, FILTER_SIZE* sizeof(unsigned long long int), hipMemcpyHostToDevice);
        if(cuda_err != hipSuccess || cuda_err2 != hipSuccess || cuda_err3 != hipSuccess)
        {
            printf("Failed with error part2 %s \n", hipGetErrorString(cuda_err));
        }


        for (int i = 2; i >= 0; i--) //R G B channel respectively
        {
            cuda_gaussian_filter<<<(resolution) / TILE_WIDTH, TILE_WIDTH>>>(cuda_input_image, cuda_output_image, img_width, img_height, i, cuda_filter_G, (int)sqrt((int)FILTER_SIZE), FILTER_SCALE, resolution);
            cuda_err = hipDeviceSynchronize();

            if(cuda_err != hipSuccess)
            {
                printf("Failed with error part3 %s \n", hipGetErrorString(cuda_err));
                return -1;
            }
        }

        // copy memory from GPU to host
        hipMemcpy(output_image, cuda_output_image, resolution * sizeof(unsigned char), hipMemcpyDeviceToHost);
        //---------------------CUDA main part-------------------------//

        // write output BMP file
        outputblur_name = inputfile_name.substr(0, inputfile_name.size() - 4)+ "_blur_cuda.bmp";
        bmpReader->WriteBMP(outputblur_name.c_str(), img_width, img_height, output_image);
        // free memory space
        free(input_image);
        free(output_image);
        hipFree(cuda_input_image);
        hipFree(cuda_output_image);
        hipFree(cuda_filter_G);
    }

    // diff pic if needed
    /* 
    printf("diff img \n");

    string inputfile_name2 = inputfile_name.substr(0, inputfile_name.size() - 4)+ "_blur.bmp";
    unsigned char* input_image2 = bmpReader -> ReadBMP(inputfile_name2.c_str(), &img_width, &img_height);

    string inputfile_name3 = inputfile_name.substr(0, inputfile_name.size() - 4)+ "_blur_cuda.bmp";
    unsigned char* input_image3 = bmpReader -> ReadBMP(inputfile_name3.c_str(), &img_width, &img_height);
    cout << "name 2 3 " << inputfile_name2 << " , " << inputfile_name3 << endl;
    for (int j = 0; j < img_width * img_height * 3; j+=3)
    {
        if(input_image2[j] != input_image3[j])
        {
            printf("Normal %d, %d, %d Dim %d, %d, %d \n", input_image2[j], input_image2[j + 1], input_image2[j +2], input_image3[j], input_image3[j + 1], input_image3[j +2]);
        }
    }
    */
    

    return 0;
}
